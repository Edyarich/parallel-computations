#include "hip/hip_runtime.h"
#include "KernelAdd.cuh"

int main(int argc, char** argv) {
    assert(argc == 3 && "Wrong arguments count");
    int blockSize = static_cast<int>(std::strtol(argv[1], nullptr, 10));
    int N = static_cast<int>(std::strtol(argv[2], nullptr, 10));
    int numBlocks = fmin(maxXBlocks, (N + blockSize - 1) / blockSize);

    size_t size = N * sizeof(float);
    float *x = (float*)calloc(N, sizeof(float));
    float *y = (float*)calloc(N, sizeof(float));
    float *res = (float*)calloc(N, sizeof(float));

    float *d_x = nullptr;
    float *d_y = nullptr;
    float *d_res = nullptr;

    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);
    hipMalloc(&d_res, size);

    for (int i = 0; i < N; ++i) {
        x[i] = i;
        y[i] = -i;
    }

    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    hipEvent_t start;
    hipEvent_t stop;

    // Creating event
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    KernelAdd<<<numBlocks, blockSize>>>(N, d_x, d_y, d_res);

    // hipEventRecord(stop);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(res, d_res, size, hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    float maxError = CheckCalculation(N, res, 0.0f);
    std::cout << "maxError = " << maxError << std::endl;
    std::cout << "Elapsed Time = " << milliseconds << std::endl;
    WriteToFile(OUT_FILENAME, N, blockSize, milliseconds);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);

    free(x);
    free(y);
    free(res);
    return 0;
}
