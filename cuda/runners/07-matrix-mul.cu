#include <MatrixMul.cuh>

int main(int argc, char** argv) {
    assert(argc == 5 && "Wrong arguments count");
    int blockXSize = static_cast<int>(std::strtol(argv[1], nullptr, 10));
    int aHeight = static_cast<int>(std::strtol(argv[2], nullptr, 10));
    int aWidth = static_cast<int>(std::strtol(argv[3], nullptr, 10));
    int bWidth = static_cast<int>(std::strtol(argv[4], nullptr, 10));
    int bHeight = aWidth;
    int blockYSize = blockXSize;

    size_t aN = aHeight * aWidth;
    size_t bN = bHeight * bWidth;
    size_t resN = aHeight * bWidth;

    size_t aSize = aN * sizeof(float);
    size_t bSize = bN * sizeof(float);
    size_t resSize = resN * sizeof(float);

    float *a = (float*)malloc(aSize);
    float *b = (float*)malloc(bSize);
    float *res = (float*)malloc(resSize);

    FillData(a, aN);
    FillData(b, bN, 2.0f);

    size_t a_pitch = 0;
    size_t b_pitch = 0;
    size_t res_pitch = 0;
    float *d_a = nullptr;
    float *d_b = nullptr;
    float *d_res = nullptr;

    hipMallocPitch(&d_a, &a_pitch, aWidth * sizeof(float), aHeight);
    hipMallocPitch(&d_b, &b_pitch, bWidth * sizeof(float), bHeight);
    hipMallocPitch(&d_res, &res_pitch, bWidth * sizeof(float), aHeight);

    hipMemcpy2D(d_a, a_pitch, a, aWidth * sizeof(float), aWidth * sizeof(float), aHeight,
                 hipMemcpyHostToDevice);
    hipMemcpy2D(d_b, b_pitch, b, bWidth * sizeof(float), bWidth * sizeof(float), bHeight,
                 hipMemcpyHostToDevice);

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    int numXBlocks = (res_pitch / sizeof(float) + blockXSize - 1) / blockXSize;
    int numYBlocks = (aHeight + blockYSize - 1) / blockYSize;
    int blockSizeInBytes = blockXSize * blockYSize * sizeof(float);

    dim3 numBlocks(numXBlocks, numYBlocks);
    dim3 blockSize(blockXSize, blockYSize);

    MatrixMul<<<numBlocks, blockSize, blockSizeInBytes * 2>>>(aHeight,
                                                                a_pitch / sizeof(float),
                                                                b_pitch / sizeof(float),
                                                                d_a,
                                                                d_b,
                                                                d_res);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy2D(res, bWidth * sizeof(float), d_res, res_pitch, bWidth * sizeof(float), aHeight,
                 hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // PrintMatrix(res, aHeight, bWidth);
    float maxError = CheckCalculation(resN, res, 2.0f * aWidth);
    std::cout << "maxError = " << maxError << std::endl;
    std::cout << "Elapsed Time = " << milliseconds << std::endl;
    WriteToFile(OUT_FILENAME,
                aWidth * aHeight,
                bWidth * bHeight,
                blockXSize,
                blockYSize,
                milliseconds);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);

    free(a);
    free(b);
    free(res);
    return 0;
}

