#include "hip/hip_runtime.h"
#include <KernelMatrixAdd.cuh>

int main(int argc, char** argv) {
    assert(argc == 5 && "Wrong arguments count");
    int blockXSize = static_cast<int>(std::strtol(argv[1], nullptr, 10));
    int blockYSize = static_cast<int>(std::strtol(argv[2], nullptr, 10));
    int matXSize = static_cast<int>(std::strtol(argv[3], nullptr, 10));
    int matYSize = static_cast<int>(std::strtol(argv[4], nullptr, 10));

    size_t N = matXSize * matYSize;
    size_t size = N * sizeof(float);
    
    float *x = (float*)malloc(size);
    float *y = (float*)malloc(size);
    float *res = (float*)malloc(size);

    for (int i = 0; i < N; ++i) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    size_t pitch = 0;
    float *d_x = nullptr;
    float *d_y = nullptr;
    float *d_res = nullptr;

    hipMallocPitch(&d_x, &pitch, matXSize * sizeof(float), matYSize);
    hipMallocPitch(&d_y, &pitch, matXSize * sizeof(float), matYSize);
    hipMallocPitch(&d_res, &pitch, matXSize * sizeof(float), matYSize);

    // std::cout << "Matrix width = " << pitch / sizeof(float) << std::endl;

    hipMemcpy2D(d_x, pitch, x, matXSize * sizeof(float), matXSize * sizeof(float), matYSize,
                                                                            hipMemcpyHostToDevice);
    hipMemcpy2D(d_y, pitch, y, matXSize * sizeof(float), matXSize * sizeof(float), matYSize,
                                                                            hipMemcpyHostToDevice);

    hipEvent_t start;
    hipEvent_t stop;

    // Creating event
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    int numXBlocks = fmin(maxXBlocks, (pitch / sizeof(float) + blockXSize - 1) / blockXSize);
    int numYBlocks = fmin(maxYBlocks, (matYSize + blockYSize - 1) / blockYSize);

    dim3 numBlocks(numXBlocks, numYBlocks);
    dim3 blockSize(blockXSize, blockYSize);

    KernelMatrixAdd<<<numBlocks, blockSize>>>(matYSize, pitch / sizeof(float), 0, d_x, d_y, d_res);

    // hipEventRecord(stop);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy2D(res, matXSize * sizeof(float), d_res, pitch, matXSize * sizeof(float), matYSize,
                    hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    float maxError = CheckCalculation(N, res);
    std::cout << "maxError = " << maxError << std::endl;
    std::cout << "Elapsed Time = " << milliseconds << std::endl;
    WriteToFile(OUT_FILENAME, matXSize, matYSize, blockXSize, blockYSize, milliseconds);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);

    free(x);
    free(y);
    free(res);
    return 0;
}
