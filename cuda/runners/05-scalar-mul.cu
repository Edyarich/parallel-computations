#include <ScalarMulRunner.cuh>

int main(int argc, char** argv) {
    assert(argc == 3 && "Wrong arguments count");
    int blockSize = static_cast<int>(std::strtol(argv[1], nullptr, 10));
    int N = static_cast<int>(std::strtol(argv[2], nullptr, 10));

    size_t size = N * sizeof(float);

    float *x = (float*)malloc(size);

    for (int i = 0; i < N; ++i) {
        x[i] = 1.0f;
    }

    hipEvent_t start;
    hipEvent_t stop;

    // Creating event
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    float first_scalar_mul = ScalarMulSumPlusReduction(N, x, x, blockSize);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "First scalar mul error = " << first_scalar_mul - N << std::endl;
    std::cout << "Elapsed Time = " << milliseconds << std::endl;
    WriteToFile(OUT_FIRST_FILENAME, N, blockSize, milliseconds);
    ////////////////////////////////////////////////////////////////////////////////////////////////
    hipEventRecord(start);

    float second_scalar_mul = ScalarMulTwoReductions(N, x, x, blockSize);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Second scalar mul error = " << second_scalar_mul - N << std::endl;
    std::cout << "Elapsed Time = " << milliseconds << std::endl;
    WriteToFile(OUT_SND_FILENAME, N, blockSize, milliseconds);

    free(x);
    return 0;
}


