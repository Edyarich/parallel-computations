#include <CosineVector.cuh>


int main(int argc, char** argv) {
    assert(argc == 3 && "Wrong arguments count");
    int blockSize = static_cast<int>(std::strtol(argv[1], nullptr, 10));
    int N = static_cast<int>(std::strtol(argv[2], nullptr, 10));

    size_t size = N * sizeof(float);

    float *x = (float*)malloc(size);
    float *y = (float*)malloc(size);

    for (int i = 0; i < N; ++i) {
        x[i] = 1.0f;
        y[i] = -1.0f;
    }

    hipEvent_t start;
    hipEvent_t stop;

    // Creating event
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    float cos_angle = CosineVector(N, x, y, blockSize);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Cosine angle error = " << cos_angle + 1.0f << std::endl;
    std::cout << "Elapsed Time = " << milliseconds << std::endl;
    WriteToFile(OUT_FILENAME6, N, blockSize, milliseconds);

    free(x);
    free(y);
    return 0;

}

