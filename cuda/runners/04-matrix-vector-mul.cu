#include "hip/hip_runtime.h"
#include <MatrixVectorMul.cuh>

int main(int argc, char** argv) {
    assert(argc == 4 && "Wrong arguments count");
    int blockSize = static_cast<int>(std::strtol(argv[1], nullptr, 10));
    int matXSize = static_cast<int>(std::strtol(argv[2], nullptr, 10));
    int matYSize = static_cast<int>(std::strtol(argv[3], nullptr, 10));

    size_t N = matXSize * matYSize;

    float *mat = (float*)malloc(N * sizeof(float));
    float *vec = (float*)malloc(matXSize * sizeof(float));
    float *res = (float*)malloc(matYSize * sizeof(float));

    FillData(mat, N);
    FillData(vec, matXSize);

    size_t pitch = 0;
    float *d_mat = nullptr;
    float *d_vec = nullptr;
    float *d_res = nullptr;

    hipMallocPitch(&d_mat, &pitch, matXSize * sizeof(float), matYSize);
    hipMallocPitch(&d_vec, &pitch, matXSize * sizeof(float), 1);
    hipMalloc(&d_res, matYSize * sizeof(float));

    hipMemcpy2D(d_mat, pitch, mat, matXSize * sizeof(float), matXSize * sizeof(float), matYSize,
                 hipMemcpyHostToDevice);
    hipMemcpy2D(d_vec, pitch, vec, matXSize * sizeof(float), matXSize * sizeof(float), 1,
                 hipMemcpyHostToDevice);

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    int numBlocks = fmin(maxXBlocks, (matYSize + blockSize - 1) / blockSize);
    int sharedMemInBytes = blockSize  * sizeof(float);

    hipEventRecord(start);

    MatrixVectorMul<<<numBlocks, blockSize, sharedMemInBytes>>>(matYSize,
                                                                pitch / sizeof(float),
                                                                d_mat,
                                                                d_vec,
                                                                d_res);

    // hipEventRecord(stop);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(res, d_res, matYSize * sizeof(float), hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // PrintMatrix(res, 1, matYSize);
    float maxError = CheckCalculation(matYSize, res, matXSize);
    std::cout << "maxError = " << maxError << std::endl;
    std::cout << "Elapsed Time = " << milliseconds << std::endl;
    WriteToFile(OUT_FILENAME, matXSize, matYSize, blockSize, 1, milliseconds);

    hipFree(d_mat);
    hipFree(d_vec);
    hipFree(d_res);

    free(mat);
    free(vec);
    free(res);
    return 0;
}
