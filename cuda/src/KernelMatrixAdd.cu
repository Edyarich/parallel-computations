#include "hip/hip_runtime.h"
#include <KernelMatrixAdd.cuh>

__global__ void KernelMatrixAdd(int height, int width, int pitch,
                                    float* A, float* B, float* result) {
    int width_ind = blockIdx.x * blockDim.x + threadIdx.x;
    int height_ind = blockIdx.y * blockDim.y + threadIdx.y;

    int width_stride = blockDim.x * gridDim.x;
    int height_stride = blockDim.y * gridDim.y;

    for (int i = height_ind; i < height; i += height_stride) {
        for (int j = width_ind; j < width; j += width_stride) {
            int idx = i * width + j;
            result[idx] = A[idx] + B[idx];
        }
    }
}
