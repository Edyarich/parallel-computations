#include "hip/hip_runtime.h"
#include <CommonKernels.cuh>


__global__ void ReduceSum(int numElements, float* input, float* output) {
    extern __shared__ float sh_data[];
    int tid = blockDim.x * blockIdx.x * 2 + threadIdx.x;
    int local_tid = threadIdx.x;

    if (tid + blockDim.x < numElements) {
        sh_data[local_tid] = input[tid] + input[tid + blockDim.x];
    } else if (tid < numElements) {
        sh_data[local_tid] = input[tid];
    } else {
        sh_data[local_tid] = 0;
    }
    __syncthreads();

    for (int step = blockDim.x / 2; step >= 1; step >>= 1) {
        if (threadIdx.x < step) {
            sh_data[local_tid] += sh_data[local_tid + step];
            __syncthreads();
        } else {
            break;
        }
    }

    if (threadIdx.x == 0) {
        output[blockIdx.x] = sh_data[0];
    }
}

__global__ void SumBlocks(int numElements, float* input, float* result) {
    int local_tid = threadIdx.x;
    float thread_sum = 0.0f;

    for (int i = local_tid; i < numElements; i += blockDim.x) {
        thread_sum += input[i];
    }
    
    result[local_tid] = thread_sum;
}

float Sum(int numElements, float* vector, int blockSize) {
    int numBlocks = (numElements + blockSize - 1) / blockSize;
    float* result = (float*)calloc(blockSize, sizeof(float));

    float* d_vec = nullptr;
    float* d_result = nullptr;

    hipMalloc(&d_vec, numElements * sizeof(float));
    hipMalloc(&d_result, numBlocks * sizeof(float));

    hipMemcpy(d_vec, vector, numElements * sizeof(float), hipMemcpyHostToDevice);

    int remainderSize = numElements;

    while (numBlocks > 1) {
        ReduceSum<<<numBlocks, blockSize, blockSize * sizeof(float)>>>(remainderSize,
                                                                       d_vec,
                                                                       d_result);
        hipMemcpy(d_vec, d_result, numBlocks * sizeof(float), hipMemcpyDeviceToDevice);

        remainderSize = numBlocks;
        numBlocks = (numBlocks + blockSize - 1) / blockSize;
    }

    hipMemcpy(result, d_vec, remainderSize * sizeof(float), hipMemcpyDeviceToHost);

    float sum = 0.0;
    for (int i = 0; i < remainderSize; ++i) {
        sum += result[i];
    }

    hipFree(d_vec);
    hipFree(d_result);
    free(result);

    return sum;
}

void WriteToFile(const char* filename, const int data_size,
                    const int block_size, const double time) {
    std::fstream outfile;
    outfile.open(filename, std::ios::out | std::ios::app);
    outfile << data_size << ' ' << block_size << ' ' << time << '\n';
    outfile.close();
}

void WriteToFile(const char* filename, const int data_x_size, const int data_y_size,
                    const int block_x_size, const int block_y_size, const double time) {
    std::fstream outfile;
    outfile.open(filename, std::ios::out | std::ios::app);
    outfile << data_x_size << ' ' << data_y_size << ' ' << block_x_size << ' ';
    outfile << block_y_size << ' ' << time << '\n';
    outfile.close();
}

float CheckCalculation(int numElements, float* result, float true_value) {
    float maxError = 0.0f;
    for (int i = 0; i < numElements; ++i) {
        maxError = fmax(maxError, fabs(result[i] - true_value));
    }
    return maxError;
}

void FillData(float* data, int size, float value) {
    for (int i = 0; i < size; ++i) {
        data[i] = value;
    }
}

void PrintMatrix(float* matrix, int height, int width) {
    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < width; ++j) {
            std::cout << i << " " << j << " " << matrix[i * width + j] << "\n";
        }
    }
}

void PrintArray(float* array, int size) {
    for (int i = 0; i < size; ++i) {
        std::cout << i << " " << array[i] << std::endl;
    }
}