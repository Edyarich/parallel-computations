#include "hip/hip_runtime.h"
#include <MatrixVectorMul.cuh>

__global__
void MatrixVectorMul(int height, int width, float* matrix, float* vector, float* result) {
    extern __shared__ float sh_data[];
    float* mat_window = sh_data;

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float thread_sum = 0.0;

    for (int col = 0; col < width; ++col) {
        int mat_idx = tid * width + col;

        if (tid < height && col < width) {
            mat_window[threadIdx.x] = matrix[mat_idx];
        }

        __syncthreads();

        thread_sum += mat_window[threadIdx.x] * vector[col];
        __syncthreads();
    }

    if (tid < height) {
        result[tid] = thread_sum;
    }
}
